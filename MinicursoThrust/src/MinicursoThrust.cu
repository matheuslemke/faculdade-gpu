#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : MinicursoThrust.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <algorithm>
#include <iostream>
#include <numeric>
#include <vector>
#include <thrust/make_transform_iterator>

#include <thrust/reduce.h>
#include <thrust/device_vector.h>

using namespace std;

struct quadrado {
	__host__ __device__
	float operator()(float x) {
		return x * x;
	}
};

int main(void) {
	thrust::host_vector<int> h_vec(10);

	thrust::generate(h_vec.begin(), h_vec.end(), rand);

	thrust::device_vector<int> d_vec = h_vec;
	thrust::device_vector<int> d_vec_res(10);

	int resultado = thrust::transform_reduce(d_vec.begin(), d_vec.end(),
			quadrado(), 0.0f, thrust::plus());

	cout << sqrtf(resultado) << endl;

	return 0;
}
