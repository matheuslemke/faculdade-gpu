#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Teste.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0 / data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size) {
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void ** )&gpuData, sizeof(float) * size));
	CUDA_CHECK_RETURN(
			hipMemcpy(gpuData, data, sizeof(float) * size,
					hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>>(gpuData, size);

	CUDA_CHECK_RETURN(
			hipMemcpy(rc, gpuData, sizeof(float) * size,
					hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

float *cpuReciprocal(float *data, unsigned size) {
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt)
		rc[cnt] = 1.0 / data[cnt];
	return rc;
}

void initialize(float *data, unsigned size) {
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5 * (i + 1);
}

__global__ void addKernel(const int *a, const int *b, int *c) {
	c[blockId.x] = a[blockId.x] + b[blockId.x];
}

int main(void) {
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	const int c[arraySize] = { 0, 0, 0, 0, 0 };

	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**) &dev_a, arraySize * sizeof(int));
	hipMalloc((void**) &dev_b, arraySize * sizeof(int));
	hipMalloc((void**) &dev_c, arraySize * sizeof(int));

	hipMemcpy(dev_a, &a, arraySize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, arraySize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, &c, arraySize * sizeof(int), hipMemcpyHostToDevice);

	addKernel<<<arraySize, 1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(&c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

	int var;
	for (var = 0; var < arraySize; ++var) {
		printf("%d ", c[var]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

